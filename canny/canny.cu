#include "hip/hip_runtime.h"
#include "image_prep.h"
#include "canny.h"
#include <iostream>
#include <unistd.h>
#include <string>

// TODO: optimize multiplications!
// TODO: worry about memory locality later
// TODO: signed or unsigned? chars or shorts?
// TODO: try separable filters

hipError_t err = hipSuccess;
dim3 dimGrid, dimBlock;

#define CUDAERR(fn, msg)\
	if ((err = fn) != hipSuccess) {\
		std::cerr << "cuda error: " msg " ("\
			<< hipGetErrorString(err) << ")" << std::endl;\
		_exit(-1);\
	}

// performs a gaussian blur on an image
__host__ void blur(float blurSize, byte *dImg, byte *dImgOut)
{
        float *hFlt, *dFlt;
        unsigned fltSize;

        gaussian_filter(blurSize, &hFlt, &fltSize);

        // allocate and copy filter to device
	CUDAERR(hipMalloc((void **) &dFlt, fltSize*fltSize*sizeof(float)),
		"allocating dFlt");

	CUDAERR(hipMemcpy(dFlt, hFlt, fltSize*fltSize*sizeof(float),
		hipMemcpyHostToDevice), "copying hFlt to dFlt");

        // blur image (for testing)
        conv2d<<<dimGrid, dimBlock>>>(dImg, dFlt, dImgOut,
                height, width, fltSize, fltSize);

        // cleanup
        free(hFlt);
        CUDAERR(hipFree(dFlt), "freeing dFlt");
}

// basic sobel kernel
// out is the magnitude of the gradient
// out2 is the angle of the gradient
__global__ void sobel(byte *img, byte *out, byte *out2, int h, int w)
{
	int vKer, hKer, y, x;

	y = blockDim.y*blockIdx.y + threadIdx.y;
	x = blockDim.x*blockIdx.x + threadIdx.x;

	if (y <= 0 || y >= h-1 || x <= 0 || x >= w-1) {
		return;
	}

	vKer = img[(y-1)*w+(x-1)]*1 + img[(y-1)*w+x]*2 + img[(y-1)*w+(x+1)]*1 +
		img[(y+1)*w+(x-1)]*-1 + img[(y+1)*w+x]*-2 + img[(y+1)*w+(x+1)]*-1;

	hKer = img[(y-1)*w+(x-1)]*1 + img[(y-1)*w+(x+1)]*-1 +
		img[y*w+(x-1)]*2 + img[y*w+(x+1)]*-2 +
		img[(y+1)*w+(x-1)]*1 + img[(y+1)*w+(x+1)]*-1;

	out[y*w+x] = min(sqrtf(hKer*hKer + vKer*vKer), 255.);
	out2[y*w+x] = ((byte)roundf((atan2f(vKer, hKer)+M_PI) / (M_PI/4))) % 4;
}

// perform edge thinning
__global__ void edge_thin(byte *mag, byte *angle, byte *out, int h, int w)
{
	int y, x, y1, x1, y2, x2;

	y = blockDim.y*blockIdx.y + threadIdx.y;
	x = blockDim.x*blockIdx.x + threadIdx.x;

	if (y <= 0 || y >= h-1 || x <= 0 || x >= w-1) {
		return;
	}

	// if not greater than angles in both directions, then zero
	switch (angle[y*w + x]) {
	case 0:
		// horizontal
		y1 = y2 = y;
		x1 = x-1;
		x2 = x+1;
		break;
	case 3:
		// 135
		y1 = y-1;
		x1 = x+1;
		y2 = y+1;
		x2 = x-1;
		break;
	case 2:
		// vertical
		x1 = x2 = x;
		y1 = y-1;
		y2 = y+1;
		break;
	case 1:
		// 45
		y1 = y-1;
		x1 = x-1;
		y2 = y+1;
		x2 = x+1;
	}

	if (mag[y1*w + x1] >= mag[y*w + x] || mag[y2*w + x2] >= mag[y*w + x]) {
		out[y*w + x] = 0;
	} else {
		out[y*w + x] = mag[y*w + x];
	}
}

// perform double thresholding
__global__ void edge_thin(byte *dImg, byte *out, int h, int w, byte t1, byte t2)
{
	int y, x, ind, grad;

	y = blockDim.y*blockIdx.y + threadIdx.y;
	x = blockDim.x*blockIdx.x + threadIdx.x;

	if (y <= 0 || y >= h-1 || x <= 0 || x >= w-1) {
		return;
	}

	ind = y*w + x;
	grad = dImg[ind];
	if (grad < t1) {
		out[ind] = 0;
	} else if (grad < t2) {
		out[ind] = t2;
	} else {
		out[ind] = 255;
	}
}

// perform canny edge detection
__host__ void canny(byte *dImg, byte *dImgOut)
{
	byte *dTmp, *dImgTmp;

	CUDAERR(hipMalloc((void**)&dImgTmp, width*height), "alloc dImgTmp");

	std::cout << "Performing Gaussian blurring..." << std::endl;
	blur(1.4, dImg, dImgOut);

	std::cout << "Performing Sobel filter..." << std::endl;
	sobel<<<dimGrid, dimBlock>>>(dImgOut, dImg, dImgTmp, height, width);
	CUDAERR(hipGetLastError(), "launch sobel kernel");

	std::cout << "Performing edge thinning..." << std::endl;
	edge_thin<<<dimGrid, dimBlock>>>(dImg, dImgTmp, dImgOut, height, width);
	CUDAERR(hipGetLastError(), "launch edge thinning kernel");

	std::cout << "Performing double thresholding..." << std::endl;
	edge_thin<<<dimGrid, dimBlock>>>(dImgOut, dImgTmp, height, width,
		255*0.2, 255*0.5);
	CUDAERR(hipGetLastError(), "launch double thresholding kernel");

	// TODO: remove this
	CUDAERR(hipMemcpy(dImgOut, dImgTmp, width*height, hipMemcpyDeviceToDevice),
		"TESTING");

	// dTmp = dImg;
	// dImg = dImgOut;
	// dImgOut = dTmp;

	CUDAERR(hipFree(dImgTmp), "freeing dImgTmp");
}

__host__ int main(int argc, char **argv)
{
	std::string filename;
	unsigned i, channels, rowStride, blockSize;
	byte *hImg, *dImg, *dImgMono, *dImgMonoOut;

	// get image name
	std::cout << "Enter filename of image (*.png): ";
	std::cin >> filename;

	// get image
	std::cout << "Reading image from file..." << std::endl;
	read_png_file(const_cast<char *>(filename.c_str()));
	channels = color_type==PNG_COLOR_TYPE_RGBA ? 4 : 3;
	rowStride = width*channels;

	std::cout << "Channels: " << channels << std::endl;

	// allocate memory
	std::cout << "Allocating host and device buffers..." << std::endl;
	hImg = (byte *)malloc(width*height*channels);
	CUDAERR(hipMalloc((void **)&dImg, width*height*channels),
		"hipMalloc dImg");
	CUDAERR(hipMalloc((void **)&dImgMono, width*height),
		"hipMalloc dImgMono");
	CUDAERR(hipMalloc((void **)&dImgMonoOut, width*height),
		"hipMalloc dImgMonoOut");

	// copy image from row-pointers to device
	for (i = 0; i < height; ++i) {
		memcpy(hImg + i*rowStride, row_pointers[i], rowStride);
	}

	// copy image to device
	std::cout << "Copying image to device..." << std::endl;
	CUDAERR(hipMemcpy(dImg, hImg, width*height*channels,
		hipMemcpyHostToDevice), "hipMemcpy to device");

	// set kernel parameters (same for all future kernel invocations)
	// TODO: calculate best grid/block dim depending on the device
	blockSize = 32;
	dimGrid = dim3(ceil(rowStride*1./blockSize),
		ceil(height*1./blockSize), 1);
	dimBlock = dim3(blockSize, blockSize, 1);

	// convert to grayscale
	std::cout << "Converting to grayscale..." << std::endl;
	toGrayScale<<<dimGrid, dimBlock>>>(dImg, dImgMono, height, width,
		channels);
	CUDAERR(hipGetLastError(), "launch toGrayScale kernel");

	// canny edge detection
	std::cout << "Performing canny edge-detection..." << std::endl;
	canny(dImgMono, dImgMonoOut);

	// convert back from grayscale
	std::cout << "Convert image back to multi-channel..." << std::endl;
	fromGrayScale<<<dimGrid, dimBlock>>>(dImgMonoOut, dImg,
		height, width, channels);
	CUDAERR(hipGetLastError(), "launch fromGrayScale kernel");

	// copy image back to host
	std::cout << "Copy image back to host..." << std::endl;
	CUDAERR(hipMemcpy(hImg, dImg, width*height*channels,
		hipMemcpyDeviceToHost), "hipMemcpy to host");

	// copy image back to row_pointers
	std::cout << "Copy image back to row_pointers..." << std::endl;
	for (i = 0; i < height; ++i) {
		memcpy(row_pointers[i], hImg + i*rowStride, rowStride);
	}

	// copy image back from device
	std::cout << "Writing image back to file..." << std::endl;
	write_png_file("test.png");

	// freeing pointers
	std::cout << "Freeing device memory..." << std::endl;
	CUDAERR(hipFree(dImg), "freeing dImg");
	CUDAERR(hipFree(dImgMono), "freeing dImgMono");
	CUDAERR(hipFree(dImgMonoOut), "freeing dImgMonoOut");

	std::cout << "Done." << std::endl;
}
